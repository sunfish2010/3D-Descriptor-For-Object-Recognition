#include "hip/hip_runtime.h"
#include "search.h"


__global__ void kernSearchRadius(int N, int n, const int max_neighbor, const PointType *surface, float radius,
        const int *feature_indices, const Eigen::Vector4f inv_radius, const Eigen::Vector4i min_pi,
        int * neighbor_indices, int* num_neighbors, float* dist){
//
//    if (index < N) {
//        PointType pt = surface[index];
//        if (isfinite(pt.x) && isfinite(pt.y) && isfinite(pt.z)){
//            uint8_t i = static_cast<uint8_t >(floor(pt.x * inv_radius[0]));
//            uint8_t j = static_cast<uint8_t >(floor(pt.y * inv_radius[1]));
//            uint8_t k = static_cast<uint8_t >(floor(pt.z * inv_radius[2]));
//            float curr_dist = (pt.x - i) * (pt.x - i) + (pt.y - j) * (pt.y - j) + (pt.z - k) * (pt.z - k);
//            i -= min_pi[0];
//            j -= min_pi[1];
//            k -= min_pi[2];
//            for (int idx = 0; idx < n; ++idx ){
//                if (i >= search_range[idx] && i <= search_range[idx + 3]
//                    && j >= search_range[idx + 1] && j <= search_range[idx + 4]
//                    && k >= search_range[idx + 2] && k <= search_range[idx + 5]){
//                    while(atomicCAS(&state, 0 , 1) != 0);
//                    int offset = num_neighbors[idx];
//                    if (offset < max_neighbor){
//                        neighbor_indices[idx * max_neighbor + offset] = index;
//                        dist[idx * max_neighbor + offset] = curr_dist;
//                        atomicAdd(&num_neighbors[idx], 1);
//                    }
//                    atomicExch(&state, 0);
//
//
//
//                }
//            }
//        }
//    }
}

Search::~Search() {

//    hipFree(dev_grid_indices);
    dev_neighbor_indices = NULL;
    dev_features_indices = NULL;
    dev_pos_surface = NULL;
}

void Search::search(const Eigen::Vector4f &inv_radius,
        const Eigen::Vector4i &pc_dimension, const Eigen::Vector4i &min_p) {
    if (!_surface || !_input || !_feature_indices || _N_features > _N_surface){
        std::cerr << "Search function not properly setup" << std::endl;
        exit(1);
    }

    if (_method == SearchMethod::KDTree) {
        std::cout << "Function not implemented yet" << std::endl;
        exit(1);
    }
    if (_method == SearchMethod::Radius){
        if(_radius == 0) {
            std::cerr << "Search function not properly setup" << std::endl;
            exit(1);
        }

        // computing indices to search

        hipMalloc((void**)&dev_features_indices, _N_features * sizeof(int));
        checkCUDAError("mallod dev_features_indices error");
        hipMemcpy(dev_features_indices, &(*_feature_indices)[0], _N_features * sizeof(int), hipMemcpyHostToDevice);
        checkCUDAError("memcpy dev_features_indices error");
        hipMalloc((void**)&dev_neighbor_indices, _N_features * _n * sizeof(int));
        checkCUDAError("malloc dev_neighbor indices error");
        hipMemset(dev_neighbor_indices, -1, _N_features * _n * sizeof(int));
        checkCUDAError("memset ni error");
        hipMalloc((void**)&dev_distances, _N_features * _n * sizeof(int));
        checkCUDAError("malloc dev_neighbor distances error");

        hipMalloc((void**)&dev_pos_surface, _N_surface * sizeof(PointType));
        checkCUDAError("malloc dps error");
        hipMemcpy(dev_pos_surface, &(_surface->points[0]), _N_surface * sizeof(PointType), hipMemcpyHostToDevice);
        checkCUDAError("memcpy ps error");
//        hipMalloc((void**)&dev_grid_indices,_N_surface * sizeof(int));
//        checkCUDAError("malloc gi failed");
//        hipMemcpy(dev_grid_indices, &(*_grid_indices)[0], _N_surface * sizeof(int), hipMemcpyHostToDevice);
//        checkCUDAError("memcpy gi failed");
        hipMalloc((void**)&dev_num_neighbors, _N_features * sizeof(int));
        checkCUDAError("malloc num neighbors error");
        hipMemset(dev_num_neighbors, 0, sizeof(int) * _N_features);
        checkCUDAError("memset num neighbors error");

        dim3 fullBlockPerGrid_points (static_cast<u_int32_t >((_N_surface + blockSize - 1)/blockSize));

        kernSearchRadius<<<fullBlockPerGrid_points, blockSize, _N_features * sizeof(u_int8_t) * 6>>> (_N_surface, _N_features,
                _n, dev_pos_surface, _radius, dev_features_indices, inv_radius, min_p, dev_neighbor_indices, dev_num_neighbors,
                dev_distances);
        checkCUDAError("KernSearchRadius error");

        _num_neighbors.resize(_N_features);
        hipMemcpy(&_num_neighbors[0], dev_num_neighbors, sizeof(int) * _N_features, hipMemcpyDeviceToHost);
        checkCUDAError("cudamemcpy  num neigbors issue");

        _neighbor_indices.resize(_n * _N_features);
        hipMemcpy(&_neighbor_indices[0], dev_neighbor_indices, sizeof(int) * _N_features * _n, hipMemcpyDeviceToHost);
        checkCUDAError("cudamemcpy  num neigbors issue");

        _neighbor_distances.resize(_n * _N_features);
        hipMemcpy(&_neighbor_distances[0], dev_distances, sizeof(float) * _N_features * _n, hipMemcpyDeviceToHost);
        checkCUDAError("cudamemcpy  distances issue");

        for(auto&n: _num_neighbors)
            std::cout << n << std::endl;

        hipFree(dev_pos_surface);
        hipFree(dev_neighbor_indices);
        hipFree(dev_features_indices);
        hipFree(dev_num_neighbors);

    }

}