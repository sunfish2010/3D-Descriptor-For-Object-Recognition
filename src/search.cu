#include "hip/hip_runtime.h"
#include "search.h"


__global__ void kernSearchRadius(int N, int *feature_indices, int * output, PointType *surface) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index < N){

    }
}

Search::~Search() {
    hipFree(dev_pos_surface);
    hipFree(dev_neighbor_indices);
    hipFree(dev_features_indices);
    hipFree(dev_grid_indices);
    dev_neighbor_indices = NULL;
    dev_features_indices = NULL;
    dev_pos_surface = NULL;
}

void Search::search(const pcl::PointCloud<PointType>::Ptr &output) {
    if (_method == SearchMethod::KDTree) {
        std::cout << "Function not implemented yet" << std::endl;
        exit(1);
    }
    if (_method == SearchMethod::Radius){
        hipMalloc((void**)&dev_features_indices, _N_features * sizeof(int));
        checkCUDAError("mallod dev_features_indices error");
        hipMemcpy(dev_features_indices, &(*_feature_indices)[0], _N_features * sizeof(int), hipMemcpyHostToDevice);
        checkCUDAError("memcpy dev_features_indices error");
        hipMalloc((void**)&dev_neighbor_indices, _N_features * _n * sizeof(int));
        checkCUDAError("malloc dev_neighbor indices error");
        hipMemset(dev_neighbor_indices, -1, _N_features * _n * sizeof(int));
        checkCUDAError("memset ni error");
        hipMalloc((void**)&dev_pos_surface, _N_surface * sizeof(PointType));
        checkCUDAError("malloc dps error");
        hipMemcpy(dev_pos_surface, &(_surface->points[0]), _N_surface * sizeof(PointType), hipMemcpyHostToDevice);
        checkCUDAError("memcpy ps error");
        hipMalloc((void**)&dev_grid_indices,_N_surface * sizeof(int));
        checkCUDAError("malloc gi failed");
        hipMemcpy(dev_grid_indices, &(*_grid_indices)[0], _N_surface * sizeof(int), hipMemcpyHostToDevice);
        checkCUDAError("memcpy gi failed");


    }

}