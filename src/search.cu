#include "hip/hip_runtime.h"
#include "search.h"


void KDTree::make_tree(const std::vector<pcl::SHOT352, Eigen::aligned_allocator<pcl::SHOT352>>& input) {

    std::vector<int> indices(input.size());
    std::iota(indices.begin(), indices.end(), 0);

    Node root;
    root.axis = 0;
    root.search_begin = indices.begin();
    root.search_end = indices.end();

    std::vector<Node, Eigen::aligned_allocator<Node>>Nodes;
    Nodes.emplace_back(root);

    while(!Nodes.empty()){
        Node curr = Nodes.back();
        Nodes.pop_back();
        if(curr.search_end > curr.search_begin + 1){
            curr.id = _num_elements++;
            if (curr.parent != -1){
                if (curr.isleft)
                    tree[curr.parent].left = curr.id;
                else
                    tree[curr.parent].right = curr.id;
            }

            Node left, right;
            _axis = curr.axis

            std::sort(curr.search_begin, curr.search_end,
                      [&input](size_t i1, size_t i2){
                          return input[i1].descriptor[kdtree->_axis] < input[i2].descriptor[kdtree->_axis];});

            auto mid = curr.search_begin + (curr.search_end - curr.search_begin)/2;
            curr.idx = *mid;

            if (mid - curr.search_begin > 0){
                left.id = mid - curr.search_begin+ curr.id + 1;
                left.parent = curr.id;
                left.axis = (curr.axis + 1) % _dim;
                left.isleft = true;
            }
            if (curr.search_end - mid > 1){
                right.axis = (curr.axis + 1) % _dim;
                right.id = curr.id + 1;
                right.parent = curr.id;
            }

            curr.left = left.id;
            curr.right = right.id;

            left.search_begin = curr.search_begin;
            left.search_end = mid;
            right.search_begin = mid + 1;
            right.search_end = curr.search_end;

            if (left.search_end > left.search_begin )
                Nodes.emplace_back(left);
            if (right.search_end > right.search_begin)
                Nodes.emplace_back(right);
            tree.emplace_back(curr);

        }
        else if (curr.search_begin +1 == curr.search_end){
            curr.id = _num_elements++;
            if (curr.parent != -1){
                if (curr.isleft)
                    tree[curr.parent].left = curr.id;
                else
                    tree[curr.parent].right = curr.id;
            }
            curr.idx = *curr.search_begin;
            tree.emplace_back(curr);
        }
    }


}

__device__ float descriptorDistance(const pcl::SHOT352& pt1, const pcl::SHOT352 &pt2){
    const int desclen_ = 352;
    float dist = 0;
    for (int i = 0; i < desclen_; ++i){
        float delta = pt1.descriptor[i] - pt2.descriptor[i];
        dist += delta * delta;
    }
    return sqrt(dist);
}


__global__ void kernFindCorrespondence(int N, const Node* nodes, const pcl::SHOT352* input, const pcl::SHOT352* queries,
        int* indices, float* dist){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N){
        pcl::SHOT352 query = queries[index];
        int n_idx = nodes[0].idx;
        int n_closest = 0;
        int split_axis = nodes[0].axis;
        float d_closest = descriptorDistance( input[n_idx], query);
        int curr_node = query.descriptor[split_axis] > input[n_idx].descriptor[split_axis] ?
                nodes[0].right:nodes[0].left;
        bool explored = false;
        while(true){
            while(curr_node != -1){
                n_idx = nodes[curr_node].idx;
                split_axis = nodes[curr_node].axis;
                float distance = descriptorDistance(input[n_idx], query);
                if (distance < d_closest){
                    d_closest = distance;
                    n_closest = curr_node;
                    explored = false;
                }
                curr_node = query.descriptor[split_axis] > input[n_idx].descriptor[split_axis]?
                        nodes[curr_node].right: nodes[curr_node].left;
            }
            if (explored ||node[n_closest].parent == -1){
                break;
            } else{
                // explore parents
                curr_node = node[n_closest].parent;
                n_idx = nodes[curr_node].idx;
                split_axis = nodes[curr_node].axis;
                float hyper_dist = query.descriptor[split_axis] - input[n_idx].descriptor[split_axis];
                if (abs(hyper_dist) < d_closest){
                    explored = true;
                    curr_node = hyper_dist > 0? nodes[curr_node].right:nodes[curr_node].left;
                }else{
                    break;
                }
            }
        }
        indices[index] = nodes[n_closest].idx;

    }
}


Search::~Search() {

//    hipFree(dev_grid_indices);
    dev_neighbor_indices = NULL;

}


void Search::setInputCloud(const pcl::PointCloud<pcl::SHOT352>::ConstPtr &input) {
    _input = input;
    _N_input = static_cast<int>(input->points.size());
    _kdtree.make_tree(input->points);

}

void Search::search(const pcl::CorrespondencesPtr &model_scene_corrs) {
    if (!_search || !_input || _N_input > _N_search){
        std::cerr << "Search function not properly setup" << std::endl;
        exit(1);
    }

    const std::vector<Node, Eigen::aligned_allocator<Node>>& tree = _kdtree.getTree();
    assert(_N_input == tree.size());
    Node* dev_tree = NULL;
    hipMalloc((void**)&dev_tree, _N_input * sizeof(Node));
    checkCUDAError("cudamalloc dev tree error");
    hipMemcpy(dev_tree, &tree[0], _N_input * sizeof(Node), hipMemcpyDeviceToHost);
    checkCUDAError("cudammcpy dev_tree error");

    hipMalloc((void**)&dev_neighbor_indices, _N_search * sizeof(int));
    checkCUDAError("malloc dev_neighbor indices error");
    hipMemset(dev_neighbor_indices, -1, _N_search * sizeof(int));
    checkCUDAError("memset ni error");

    hipMalloc((void**)&dev_input, _N_input * sizeof(pcl::SHOT352));
    checkCUDAError("malloc dev_neighbor distances error");
    hipMemcpy(dev_input, &(_input->points[0]), _N_input * sizeof(pcl::SHOT352), hipMemcpyHostToDevice);

    hipMalloc((void**)&dev_search, _N_search * sizeof(pcl::SHOT352));
    checkCUDAError("malloc dps error");
    hipMemcpy(dev_search, &(_search->points[0]), _N_search * sizeof(pcl::SHOT352), hipMemcpyHostToDevice);
    checkCUDAError("memcpy ps error");

    float *dev_dist = NULL;
    hipMalloc((void**)&dev_dist, _N_search * sizeof(float) );
    checkCUDAError("dev_dist malloc");


    dim3 fullBlockPerGrid_points (static_cast<u_int32_t >((_N_search + blockSize - 1)/blockSize));

//    kernSearchCorrespondence

    checkCUDAError("KernSearchCorres error");


    _neighbor_indices.resize(_N_search);
    hipMemcpy(&_neighbor_indices[0], dev_neighbor_indices, sizeof(int) * _N_search, hipMemcpyDeviceToHost);
    checkCUDAError("cudamemcpy  num neigbors issue");


    for(auto&n: _neighbor_indices)
        std::cout << n << std::endl;

    hipFree(dev_search);
    hipFree(dev_input);
    hipFree(dev_neighbor_indices);


}