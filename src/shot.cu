#include "hip/hip_runtime.h"
#include "shot.h"
#include "shot_lrf.h"

/**  Reference
 *   - F. Tombari, S. Salti, L. Di Stefano
 *     Unique Signatures of Histograms for Local Surface Description.
 *     In Proceedings of the 11th European Conference on Computer Vision (ECCV),
         *     Heraklion, Greece, September 5-11 2010.
 *   - F. Tombari, S. Salti, L. Di Stefano
 *     A Combined Texture-Shape Descriptor For Enhanced 3D Feature Matching.
 *     In Proceedings of the 18th International Conference on Image Processing (ICIP),
        *     Brussels, Belgium, September 11-14 2011.
 *
 * \author Samuele Salti, Federico Tombari
 *
 *
 * Implemented based on PCL & Original Paper's Author's implementation, modified to run on CUDA
 *
 *
 */

//
//__global__ void computeBinDistShape(int N,const pcl::Normal* norms, const pcl::ReferenceFrame *lrf,
//        double *bin_dist, int* neighbor_indices, const int n_bin, const int k){
//    int index = threadIdx.x + blockIdx.x * blockDim.x;
//    if (index < N){
//        for (int i = 0; i < k; ++i){
//            if (neighbor_indices[index * k + i] != -1){
//                const pcl::Normal& norm = norms[neighbor_indices[index * k + i]];
//                if (! isfinite(norm.normal_x) || !isfinite(norm.normal_y) || !isfinite(norm.normal_z)){
//                    bin_dist[index * k + i] = NAN;
//                }else{
//                    double cosDesc = norm.normal_x * lrf[index].z_axis[0] +
//                                     norm.normal_y * lrf[index].z_axis[1] + norm.normal_z * lrf[index].z_axis[2];
//                    if (cosDesc > 1) cosDesc = 1;
//                    else if (cosDesc < -1) cosDesc = -1;
//                    bin_dist[index * k + i] = ((1.0 + cosDesc) * n_bin) / 2;
//                }
//            }
//            else
//                bin_dist[index * k + i] = NAN;
//
//        }
//    }
//}

__device__ void rgb2lab(const float* LUT, const unsigned char r, const unsigned char g, const unsigned char b, float &a, float &b2, float &l){
    float x = (LUT[r] * 0.412453f + LUT[g] * 0.357580f + LUT[b] * 0.180423f) / 0.95047f;
    float y = LUT[r] * 0.212671f + LUT[g] * 0.715160f + LUT[b] * 0.072169f;
    float z = (LUT[r] * 0.019334f + LUT[g] * 0.119193f + LUT[b] * 0.950227f) / 1.08883f;

    x = LUT[int(x*4000) + 256];
    y = LUT[int(y*4000) + 256];
    z = LUT[int(z*4000) + 256];

    l = 116.0f * y - 16.0f;
    if (l > 100)
        l = 100.0f;

    a = 500.0f * (x - y);
    if (a > 120)
        a = 120.0f;
    else if (a <- 120)
        a = -120.0f;

    b2 = 200.0f * (y - z);
    if (b2 > 120)
        b2 = 120.0f;
    else if (b2< -120)
        b2 = -120.0f;

}

//__global__ void computeBinColorShape(int N, const PointType* surface, double *bin_dist, const int* neighbor_indices,
//        const int k, const int n_color_bin){
//    // even if the same computation is performed many times, it should still be faster that global memory
//     __shared__ float LUT[256 + 4000];
//    int index = threadIdx.x + blockIdx.x * blockDim.x;
//    int num = (4000 + 256)/blockSize + 1;
//    for (int i = 0; i < num; i++){
//        int idx = num * threadIdx.x + i;
//        if (idx  < 4000 + 256){
//            if (idx < 256){
//                float f = static_cast<float>(idx)/ 255.f;
//                if (f > 0.04045)
//                    LUT[idx] = powf((f + 0.055f)/1.055f, 2.4f);
//                else
//                    LUT[idx] = f / 12.92f;
//            }else{
//                float f = static_cast<float>(idx) / 4000.f;
//                if (f > 0.008856)
//                    LUT[idx] = powf(f, 0.3333f);
//                else
//                    LUT[idx] = (7.787f * f) + (16.f / 116.f);
//            }
//        }
//    }
//    __syncthreads();
//
//    if (index < N){
//        float L,A,B;
//        rgb2lab(LUT, surface[index].r, surface[index].g, surface[index].b, A, B, L );
//        for (int i = 0; i < k; ++i){
//            if (neighbor_indices[index * k + i] != -1){
//                float l, a, b;
//                int neighbor = neighbor_indices[index * k + i];
//                rgb2lab(LUT, surface[neighbor].r, surface[neighbor].g, surface[neighbor].b, a, b, l);
//                double color_dist = (fabs(L - l) + (fabs(A - a) + fabs(B - b))/2) / 3;
//                color_dist = color_dist > 1.0? 1.0:color_dist;
//                color_dist = color_dist < 0.0? 0.0:color_dist;
//                bin_dist[index * k + i] = color_dist * n_color_bin;
//            }else{
//                bin_dist[index * k + i] = NAN;
//            }
//        }
//    }
//}


inline __device__ bool areEquals (double val1, double val2)
{
    return (fabs (val1 - val2)<1E-15);
}


inline __device__ bool areEquals (float val1, float val2)
{
    return (fabs (val1 - val2)< 1E-8f);
}



__global__ void computeCOLORSHOT(int N, int n, const PointType *surface, const float radius, const int *feature_indices,
        const pcl::Normal* norms, const pcl::ReferenceFrame *lrf, const int n_color_bin, const int n_dist_bin,
        const int nr_grid_sector_, float* shot ){
    // even if the same computation is performed many times, it should still be faster that global memory
    __shared__ float LUT[256 + 4000];
    const double PST_PI = 3.1415926535897932384626433832795;
    const double PST_RAD_45 = 0.78539816339744830961566084581988;
    const double PST_RAD_90 = 1.5707963267948966192313216916398;
    const double PST_RAD_135 = 2.3561944901923449288469825374596;
    const double PST_RAD_180 = PST_PI;
//    const double PST_RAD_360 = 6.283185307179586476925286766558;
    const double PST_RAD_PI_7_8 = 2.7488935718910690836548129603691;
    int num = (4000 + 256)/blockSize + 1;
    for (int i = 0; i < num; i++){
        int idx = num * threadIdx.x + i;
        if (idx  < 4000 + 256){
            if (idx < 256){
                float f = static_cast<float>(idx)/ 255.f;
                if (f > 0.04045)
                    LUT[idx] = powf((f + 0.055f)/1.055f, 2.4f);
                else
                    LUT[idx] = f / 12.92f;
            }else{
                float f = static_cast<float>(idx) / 4000.f;
                if (f > 0.008856)
                    LUT[idx] = powf(f, 0.3333f);
                else
                    LUT[idx] = (7.787f * f) + (16.f / 116.f);
            }
        }
    }
    __syncthreads();
    int shapeToColorStride = nr_grid_sector_*(n_dist_bin+1);
//    double sqradius_ = radius * radius;
    double radius3_4_ = (radius * 3) / 4;
    double radius1_4_ = radius / 4;
    double radius1_2_ = radius / 2;
    const int descLength_ = 1344;
    const int maxAngularSectors_ = 32;
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n){
        int num_neighbors = 0;
        PointType central_point = surface[feature_indices[index]];
        pcl::ReferenceFrame current_frame = lrf[index];


        Eigen::Vector3f current_frame_x (current_frame.x_axis[0], current_frame.x_axis[1], current_frame.x_axis[2]);
        Eigen::Vector3f current_frame_y (current_frame.y_axis[0], current_frame.y_axis[1], current_frame.y_axis[2]);
        Eigen::Vector3f current_frame_z (current_frame.z_axis[0], current_frame.z_axis[1], current_frame.z_axis[2]);
        float L,A,B;
        rgb2lab(LUT, central_point.r, central_point.g, central_point.b, A, B, L );
        double bin_color_dist[1000];
        double bin_dist[1000];
        for(int idx = 0; idx < N;  idx++){
            PointType pt = surface[idx];
            if (isfinite(pt.x) && isfinite(pt.y) && isfinite(pt.z)) {
                if (fabs(central_point.x - pt.x) < radius  && fabs(central_point.y - pt.y) < radius
                    && fabs(central_point.z - pt.z) < radius
                    && !(pt.x == central_point.x && pt.y == central_point.y && pt.z == central_point.z)){

                    float l, a, b;
                    rgb2lab(LUT, pt.r, pt.g, pt.b, a, b, l);
                    double color_dist = (fabs(L - l) + (fabs(A - a) + fabs(B - b))/2) / 3;
                    color_dist = color_dist > 1.0? 1.0:color_dist;
                    color_dist = color_dist < 0.0? 0.0:color_dist;
                    bin_color_dist[num_neighbors] = color_dist * n_color_bin;

                    const pcl::Normal& norm = norms[idx];
                    if (! isfinite(norm.normal_x) || !isfinite(norm.normal_y) || !isfinite(norm.normal_z)){
                        bin_dist[num_neighbors] = NAN;
                    }else{
                        double cosDesc = norm.normal_x * lrf[index].z_axis[0] +
                                         norm.normal_y * lrf[index].z_axis[1] + norm.normal_z * lrf[index].z_axis[2];
                        if (cosDesc > 1) cosDesc = 1;
                        else if (cosDesc < -1) cosDesc = -1;
                        bin_dist[num_neighbors] = ((1.0 + cosDesc) * n_dist_bin) / 2;
                    }

                    int offset = descLength_ * index;
                    Eigen::Vector3f delta(pt.x- central_point.x, pt.y - central_point.y, pt.z - central_point.z);

                    double distance = sqrt((central_point.x - pt.x) * (central_point.x - pt.x) +
                        (central_point.y - pt.y) * (central_point.y - pt.y) +
                        (central_point.z - pt.z)*(central_point.z - pt.z));
                    if (areEquals (distance, 0.0))
                        continue;

                    double xInFeatRef = delta.dot (current_frame_x);
                    double yInFeatRef = delta.dot (current_frame_y);
                    double zInFeatRef = delta.dot (current_frame_z);

                    // To avoid numerical problems afterwards
                    if (fabs (yInFeatRef) < 1E-30)
                        yInFeatRef  = 0;
                    if (fabs (xInFeatRef) < 1E-30)
                        xInFeatRef  = 0;
                    if (fabs (zInFeatRef) < 1E-30)
                        zInFeatRef = 0;

                    unsigned char bit4 = ((yInFeatRef > 0) || ((yInFeatRef == 0.0) && (xInFeatRef < 0))) ? 1 : 0;
                    unsigned char bit3 = static_cast<unsigned char> (((xInFeatRef > 0) || ((xInFeatRef == 0.0)
                            && (yInFeatRef > 0))) == !bit4 );

                    assert (bit3 == 0 || bit3 == 1);

                    int desc_index = (bit4<<3) + (bit3<<2);

                    desc_index = desc_index << 1;

                    if ((xInFeatRef * yInFeatRef > 0) || (xInFeatRef == 0.0))
                        desc_index += (fabs (xInFeatRef) >= fabs (yInFeatRef)) ? 0 : 4;
                    else
                        desc_index += (fabs (xInFeatRef) > fabs (yInFeatRef)) ? 4 : 0;

                    desc_index += zInFeatRef > 0 ? 1 : 0;

                    // 2 RADII
                    desc_index += (distance > radius1_2_) ? 2 : 0;

                    int step_index_shape = static_cast<int>(floor (bin_dist[num_neighbors] +0.5));
                    int step_index_color = static_cast<int>(floor (bin_color_dist[num_neighbors] +0.5));

                    int volume_index_shape = desc_index * (n_dist_bin+1);
                    int volume_index_color = shapeToColorStride + desc_index * (n_color_bin+1);

                    //Interpolation on the cosine (adjacent bins in the histrogram)
                    bin_dist[num_neighbors] -= step_index_shape;
                    bin_color_dist[num_neighbors] -= step_index_color;

                    double intWeightShape = (1- fabs (bin_dist[num_neighbors]));
                    double intWeightColor = (1- fabs (bin_color_dist[num_neighbors]));

                    if (bin_dist[num_neighbors] > 0)
                        shot[offset + volume_index_shape + ((step_index_shape + 1) % n_dist_bin)] +=
                                static_cast<float> (bin_dist[num_neighbors]);
                    else
                        shot[offset + volume_index_shape + ((step_index_shape - 1 + n_dist_bin) % n_dist_bin)] -=
                                static_cast<float> (bin_dist[num_neighbors]);

                    if (bin_color_dist[num_neighbors] > 0)
                        shot[offset + volume_index_color + ((step_index_color+1) % n_color_bin)] +=
                                static_cast<float> (bin_color_dist[num_neighbors]);
                    else
                        shot[offset + volume_index_color + ((step_index_color - 1 + n_color_bin) % n_color_bin)] -=
                                static_cast<float> (bin_color_dist[num_neighbors]);

                    //Interpolation on the distance (adjacent husks)

                    if (distance > radius1_2_)   //external sphere
                    {
                        double radiusDistance = (distance - radius3_4_) / radius1_2_;

                        if (distance > radius3_4_) //most external sector, votes only for itself
                        {
                            intWeightShape += 1 - radiusDistance; //weight=1-d
                            intWeightColor += 1 - radiusDistance; //weight=1-d
                        }
                        else  //3/4 of radius, votes also for the internal sphere
                        {
                            intWeightShape += 1 + radiusDistance;
                            intWeightColor += 1 + radiusDistance;
                            shot[offset + (desc_index - 2) * (n_dist_bin+1) + step_index_shape] -=
                                    static_cast<float> (radiusDistance);
                            shot[offset + shapeToColorStride + (desc_index - 2) * (n_color_bin+1) + step_index_color] -=
                                    static_cast<float> (radiusDistance);
                        }
                    }
                    else    //internal sphere
                    {
                        double radiusDistance = (distance - radius1_4_) / radius1_2_;

                        if (distance < radius1_4_) //most internal sector, votes only for itself
                        {
                            intWeightShape += 1 + radiusDistance;
                            intWeightColor += 1 + radiusDistance;
                        }
                        else  //3/4 of radius, votes also for the external sphere
                        {
                            intWeightShape += 1 - radiusDistance;
                            intWeightColor += 1 - radiusDistance;
                            shot[offset + (desc_index + 2) * (n_dist_bin+1) + step_index_shape] +=
                                    static_cast<float> (radiusDistance);
                            shot[offset + shapeToColorStride + (desc_index + 2) * (n_color_bin+1) + step_index_color] +=
                                    static_cast<float> (radiusDistance);
                        }
                    }

                    //Interpolation on the inclination (adjacent vertical volumes)
                    double inclinationCos = zInFeatRef / distance;
                    if (inclinationCos < - 1.0)
                        inclinationCos = - 1.0;
                    if (inclinationCos > 1.0)
                        inclinationCos = 1.0;

                    double inclination = acos (inclinationCos);

                    assert (inclination >= 0.0 && inclination <= PST_RAD_180);

                    if (inclination > PST_RAD_90 || (fabs (inclination - PST_RAD_90) < 1e-30 && zInFeatRef <= 0))
                    {
                        double inclinationDistance = (inclination - PST_RAD_135) / PST_RAD_90;
                        if (inclination > PST_RAD_135)
                        {
                            intWeightShape += 1 - inclinationDistance;
                            intWeightColor += 1 - inclinationDistance;
                        }
                        else
                        {
                            intWeightShape += 1 + inclinationDistance;
                            intWeightColor += 1 + inclinationDistance;
//                            assert ((desc_index + 1) * (n_dist_bin+1) + step_index_shape >= 0 && (desc_index + 1) * (n_dist_bin+1) + step_index_shape < descLength_);
//                            assert (shapeToColorStride + (desc_index + 1) * (n_color_bin+ 1) + step_index_color >= 0 && shapeToColorStride + (desc_index + 1) * (n_color_bin+1) + step_index_color < descLength_);
                            shot[offset + (desc_index + 1) * (n_dist_bin+1) + step_index_shape] -=
                                    static_cast<float> (inclinationDistance);
                            shot[offset + shapeToColorStride + (desc_index + 1) * (n_color_bin+1) + step_index_color] -=
                                    static_cast<float> (inclinationDistance);
                        }
                    }
                    else
                    {
                        double inclinationDistance = (inclination - PST_RAD_45) / PST_RAD_90;
                        if (inclination < PST_RAD_45)
                        {
                            intWeightShape += 1 + inclinationDistance;
                            intWeightColor += 1 + inclinationDistance;
                        }
                        else
                        {
                            intWeightShape += 1 - inclinationDistance;
                            intWeightColor += 1 - inclinationDistance;
//                            if (!((desc_index - 1) * (n_dist_bin+1) + step_index_shape >= 0 && (desc_index - 1) * (n_dist_bin+1) + step_index_shape < descLength_)){
//                                printf("desc_index is %d, step_index_shape is %d, n_dist_bin: %d, num_neighbors %d, bin_dist: %f  \n",
//                                        desc_index, step_index_shape, n_dist_bin, num_neighbors, bin_dist[num_neighbors]);
//                            }

//                            assert ((desc_index - 1) * (n_dist_bin+1) + step_index_shape >= 0 && (desc_index - 1) * (n_dist_bin+1) + step_index_shape < descLength_);
//                            assert (shapeToColorStride + (desc_index - 1) * (n_color_bin+ 1) + step_index_color >= 0 && shapeToColorStride + (desc_index - 1) * (n_color_bin+1) + step_index_color < descLength_);
                            shot[offset + (desc_index - 1) * (n_dist_bin+1) + step_index_shape] +=
                                    static_cast<float> (inclinationDistance);
                            shot[offset + shapeToColorStride + (desc_index - 1) * (n_color_bin+1) + step_index_color] +=
                                    static_cast<float> (inclinationDistance);
                        }
                    }

                    if (yInFeatRef != 0.0 || xInFeatRef != 0.0)
                    {
                        //Interpolation on the azimuth (adjacent horizontal volumes)
                        double azimuth = atan2 (yInFeatRef, xInFeatRef);

                        int sel = desc_index >> 2;
                        double angularSectorSpan = PST_RAD_45;
                        double angularSectorStart = - PST_RAD_PI_7_8;

                        double azimuthDistance = (azimuth - (angularSectorStart + angularSectorSpan*sel)) / angularSectorSpan;
//                        assert ((azimuthDistance < 0.5 || areEquals (azimuthDistance, 0.5)) && (azimuthDistance > - 0.5 || areEquals (azimuthDistance, - 0.5)));
                        azimuthDistance = max(- 0.5, min (azimuthDistance, 0.5));

                        if (azimuthDistance > 0)
                        {
                            intWeightShape += 1 - azimuthDistance;
                            intWeightColor += 1 - azimuthDistance;
                            int interp_index = (desc_index + 4) % maxAngularSectors_;
//                            assert (interp_index * (n_dist_bin+1) + step_index_shape >= 0 && interp_index * (n_dist_bin+1) + step_index_shape < descLength_);
//                            assert (shapeToColorStride + interp_index * (n_color_bin+1) + step_index_color >= 0 && shapeToColorStride + interp_index * (n_color_bin+1) + step_index_color < descLength_);
                            shot[offset + interp_index * (n_dist_bin+1) + step_index_shape] +=
                                    static_cast<float> (azimuthDistance);
                            shot[offset + shapeToColorStride + interp_index * (n_color_bin+1) + step_index_color] +=
                                    static_cast<float> (azimuthDistance);
                        }
                        else
                        {
                            int interp_index = (desc_index - 4 + maxAngularSectors_) % maxAngularSectors_;
                            intWeightShape += 1 + azimuthDistance;
                            intWeightColor += 1 + azimuthDistance;
//                            assert (interp_index * (n_dist_bin+1) + step_index_shape >= 0 && interp_index * (n_dist_bin+1) + step_index_shape < descLength_);
//                            assert (shapeToColorStride + interp_index * (n_color_bin+1) + step_index_color >= 0 && shapeToColorStride + interp_index * (n_color_bin+1) + step_index_color < descLength_);
                            shot[offset + interp_index * (n_dist_bin+1) + step_index_shape] -=
                                    static_cast<float> (azimuthDistance);
                            shot[offset + shapeToColorStride + interp_index * (n_color_bin+1) + step_index_color] -=
                                    static_cast<float> (azimuthDistance);
                        }
                    }

                    assert (volume_index_shape + step_index_shape >= 0 &&  volume_index_shape + step_index_shape < descLength_);
                    assert (volume_index_color + step_index_color >= 0 &&  volume_index_color + step_index_color < descLength_);
                    shot[offset + volume_index_shape + step_index_shape] += static_cast<float> (intWeightShape);
                    shot[offset + volume_index_color + step_index_color] += static_cast<float> (intWeightColor);

                    num_neighbors++;
                    if (num_neighbors >= 1000) break;
                }
            }

        }
    }

}



__global__ void computeSHOT(int N, const PointType *surface, const float radius, const int *feature_indices, const int* array_indices,
                     const int* grid_start_indices, const int* grid_end_indices, const Eigen::Vector4i grid_res,
                     const Eigen::Vector4i grid_min, const Eigen::Vector4i grid_max, const Eigen::Vector4f inv_radius,
                     const pcl::Normal* norms, const pcl::ReferenceFrame *lrf,  const int n_dist_bin, float* shot ){
    // even if the same computation is performed many times, it should still be faster that global memory
    const double PST_PI = 3.1415926535897932384626433832795;
    const double PST_RAD_45 = 0.78539816339744830961566084581988;
    const double PST_RAD_90 = 1.5707963267948966192313216916398;
    const double PST_RAD_135 = 2.3561944901923449288469825374596;
    const double PST_RAD_180 = PST_PI;
//    const double PST_RAD_360 = 6.283185307179586476925286766558;
    const double PST_RAD_PI_7_8 = 2.7488935718910690836548129603691;

//    double sqradius_ = radius * radius;
    double radius3_4_ = (radius * 3) / 4;
    double radius1_4_ = radius / 4;
    double radius1_2_ = radius / 2;
    const int descLength_ = 352;
    const int maxAngularSectors_ = 32;
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < N){
        int num_neighbors = 0;
        int offset = descLength_ * index;
        PointType central_point = surface[feature_indices[index]];
        pcl::ReferenceFrame current_frame = lrf[index];

        Eigen::Vector3f current_frame_x (current_frame.x_axis[0], current_frame.x_axis[1], current_frame.x_axis[2]);
        Eigen::Vector3f current_frame_y (current_frame.y_axis[0], current_frame.y_axis[1], current_frame.y_axis[2]);
        Eigen::Vector3f current_frame_z (current_frame.z_axis[0], current_frame.z_axis[1], current_frame.z_axis[2]);
        if(!isfinite(current_frame_x[0]) || !isfinite(current_frame_x[1]) || !isfinite(current_frame_x[2])
           || !isfinite(current_frame_y[0]) || !isfinite(current_frame_y[1]) || !isfinite(current_frame_y[2])
           || !isfinite(current_frame_z[0]) || !isfinite(current_frame_z[1]) || !isfinite(current_frame_z[2])  ){
            shot[offset] = NAN;
        }
        Eigen::Vector4i ijk(static_cast<int>(floor(central_point.x * inv_radius[0])),
                            static_cast<int>(floor(central_point.y * inv_radius[1])),
                            static_cast<int>(floor(central_point.z * inv_radius[2])), 0);
        ijk = ijk - grid_min;
        double bin_dist[1000];
        for (int i = ijk[2] - 1; i <= ijk[2] + 1; i++) {
            for (int j = ijk[1] - 1; j <= ijk[1] + 1; j++) {
                for (int k = ijk[0] - 1; k <= ijk[0] + 1; k++) {
                    int _x = imax(k, 0);
                    int _y = imax(j, 0);
                    int _z = imax(i, 0);
                    _x = imin(_x, grid_max[0]);
                    _y = imin(_y, grid_max[1]);
                    _z = imin(_z, grid_max[2]);
                    int grid_idx = _x + _y * grid_res[0] + _z * grid_res[0] * grid_res[1];
                    if (grid_start_indices[grid_idx] == -1) continue;
                    for (int s = grid_start_indices[grid_idx]; s < grid_end_indices[grid_idx]; s++){
                        int idx = array_indices[s];
                        PointType pt = surface[idx];
                        if (isfinite(pt.x) && isfinite(pt.y) && isfinite(pt.z)) {
                            Eigen::Vector3f delta(pt.x- central_point.x, pt.y - central_point.y, pt.z - central_point.z);
                            double distance = sqrt(delta.dot(delta));
                            if ( distance <= radius && !(pt.x == central_point.x && pt.y == central_point.y && pt.z == central_point.z)){

                                pcl::Normal norm = norms[idx];
                                const Eigen::Vector3f norm_vec(norm.normal_x, norm.normal_y, norm.normal_z);
                                if (! isfinite(norm_vec[0]) || !isfinite(norm_vec[1]) || !isfinite(norm_vec[2])){
                                    continue;
                                }else{
                                    double cosDesc = norm_vec.dot(current_frame_z);
                                    if (cosDesc > 1) cosDesc = 1;
                                    else if (cosDesc < -1) cosDesc = -1;
                                    bin_dist[num_neighbors] = ((1.0 + cosDesc) * n_dist_bin) / 2;
                                }


                                if (areEquals (distance, 0.0))
                                    continue;

                                double xInFeatRef = delta.dot (current_frame_x);
                                double yInFeatRef = delta.dot (current_frame_y);
                                double zInFeatRef = delta.dot (current_frame_z);

                                // To avoid numerical problems afterwards
                                if (fabs (yInFeatRef) < 1E-30)
                                    yInFeatRef  = 0;
                                if (fabs (xInFeatRef) < 1E-30)
                                    xInFeatRef  = 0;
                                if (fabs (zInFeatRef) < 1E-30)
                                    zInFeatRef = 0;

                                unsigned char bit4 = static_cast<unsigned char>(((yInFeatRef > 0) || ((yInFeatRef == 0.0) && (xInFeatRef < 0))) ? 1 : 0);
                                unsigned char bit3 = static_cast<unsigned char> (((xInFeatRef > 0) ||
                                                                                  ((xInFeatRef == 0.0) && (yInFeatRef > 0))) == (!bit4));

                                assert (bit3 == 0 || bit3 == 1);

                                int desc_index = (bit4<<3) + (bit3<<2);

                                desc_index = desc_index << 1;

                                if ((xInFeatRef * yInFeatRef > 0) || (xInFeatRef == 0.0))
                                    desc_index += (fabs (xInFeatRef) >= fabs (yInFeatRef)) ? 0 : 4;
                                else
                                    desc_index += (fabs (xInFeatRef) > fabs (yInFeatRef)) ? 4 : 0;

                                desc_index += zInFeatRef > 0 ? 1 : 0;

                                // 2 RADII
                                desc_index += (distance > radius1_2_) ? 2 : 0;

                                int step_index_shape = static_cast<int>(floor (bin_dist[num_neighbors] +0.5));

                                int volume_index_shape = desc_index * (n_dist_bin+1);


                                //Interpolation on the cosine (adjacent bins in the histrogram)
                                bin_dist[num_neighbors] -= step_index_shape;


                                double intWeightShape = (1- fabs (bin_dist[num_neighbors]));

                                if (bin_dist[num_neighbors] > 0)
                                    shot[offset + volume_index_shape + ((step_index_shape + 1) % n_dist_bin)] +=
                                            static_cast<float> (bin_dist[num_neighbors]);
                                else
                                    shot[offset + volume_index_shape + ((step_index_shape - 1 + n_dist_bin) % n_dist_bin)] -=
                                            static_cast<float> (bin_dist[num_neighbors]);

                                //Interpolation on the distance (adjacent husks)

                                if (distance > radius1_2_)   //external sphere
                                {
                                    double radiusDistance = (distance - radius3_4_) / radius1_2_;

                                    if (distance > radius3_4_) //most external sector, votes only for itself
                                    {
                                        intWeightShape += 1 - radiusDistance; //weight=1-d
                                    }
                                    else  //3/4 of radius, votes also for the internal sphere
                                    {
                                        intWeightShape += 1 + radiusDistance;
                                        shot[offset + (desc_index - 2) * (n_dist_bin+1) + step_index_shape] -=
                                                static_cast<float> (radiusDistance);
                                    }
                                }
                                else    //internal sphere
                                {
                                    double radiusDistance = (distance - radius1_4_) / radius1_2_;

                                    if (distance < radius1_4_) //most internal sector, votes only for itself
                                    {
                                        intWeightShape += 1 + radiusDistance;
                                    }
                                    else  //3/4 of radius, votes also for the external sphere
                                    {
                                        intWeightShape += 1 - radiusDistance; //weight=1-d
                                        shot[offset + (desc_index + 2) * (n_dist_bin+1) + step_index_shape] +=
                                                static_cast<float> (radiusDistance);
                                    }
                                }

                                //Interpolation on the inclination (adjacent vertical volumes)
                                double inclinationCos = zInFeatRef / distance;
                                if (inclinationCos < - 1.0)
                                    inclinationCos = - 1.0;
                                if (inclinationCos > 1.0)
                                    inclinationCos = 1.0;

                                double inclination = acos (inclinationCos);

                                assert (inclination >= 0.0 && inclination <= PST_RAD_180);

                                if (inclination > PST_RAD_90 || (fabs (inclination - PST_RAD_90) < 1e-30 && zInFeatRef <= 0))
                                {
                                    double inclinationDistance = (inclination - PST_RAD_135) / PST_RAD_90;
                                    if (inclination > PST_RAD_135)
                                    {
                                        intWeightShape += 1 - inclinationDistance;
                                    }
                                    else
                                    {
                                        intWeightShape += 1 + inclinationDistance;
//                            assert ((desc_index + 1) * (n_dist_bin+1) + step_index_shape >= 0 && (desc_index + 1) * (n_dist_bin+1) + step_index_shape < descLength_);
                                        shot[offset + (desc_index + 1) * (n_dist_bin+1) + step_index_shape] -=
                                                static_cast<float> (inclinationDistance);
                                    }
                                }
                                else
                                {
                                    double inclinationDistance = (inclination - PST_RAD_45) / PST_RAD_90;
                                    if (inclination < PST_RAD_45)
                                    {
                                        intWeightShape += 1 + inclinationDistance;
                                    }
                                    else
                                    {
                                        intWeightShape += 1 - inclinationDistance;
//                            assert ((desc_index - 1) * (n_dist_bin+1) + step_index_shape >= 0 && (desc_index - 1) * (n_dist_bin+1) + step_index_shape < descLength_);
                                        shot[offset + (desc_index - 1) * (n_dist_bin+1) + step_index_shape] +=
                                                static_cast<float> (inclinationDistance);
                                    }
                                }

                                if (yInFeatRef != 0.0 || xInFeatRef != 0.0)
                                {
                                    //Interpolation on the azimuth (adjacent horizontal volumes)
                                    double azimuth = atan2 (yInFeatRef, xInFeatRef);

                                    int sel = desc_index >> 2;
                                    double angularSectorSpan = PST_RAD_45;
                                    double angularSectorStart = - PST_RAD_PI_7_8;

                                    double azimuthDistance = (azimuth - (angularSectorStart + angularSectorSpan*sel)) / angularSectorSpan;
//                        assert ((azimuthDistance < 0.5 || areEquals (azimuthDistance, 0.5)) && (azimuthDistance > - 0.5 || areEquals (azimuthDistance, - 0.5)));
                                    azimuthDistance = max(- 0.5, min (azimuthDistance, 0.5));

                                    if (azimuthDistance > 0)
                                    {
                                        intWeightShape += 1 - azimuthDistance;
                                        int interp_index = (desc_index + 4) % maxAngularSectors_;
//                            assert (interp_index * (n_dist_bin+1) + step_index_shape >= 0 && interp_index * (n_dist_bin+1) + step_index_shape < descLength_);
                                        shot[offset + interp_index * (n_dist_bin+1) + step_index_shape] +=
                                                static_cast<float> (azimuthDistance);
                                    }
                                    else
                                    {
                                        int interp_index = (desc_index - 4 + maxAngularSectors_) % maxAngularSectors_;
                                        intWeightShape += 1 + azimuthDistance;
//                            assert (interp_index * (n_dist_bin+1) + step_index_shape >= 0 && interp_index * (n_dist_bin+1) + step_index_shape < descLength_);
                                        shot[offset + interp_index * (n_dist_bin+1) + step_index_shape] -=
                                                static_cast<float> (azimuthDistance);
                                    }
                                }

//                    assert (volume_index_shape + step_index_shape >= 0 &&  volume_index_shape + step_index_shape < descLength_);
                                shot[offset + volume_index_shape + step_index_shape] += static_cast<float> (intWeightShape);

                                num_neighbors++;
                                if (num_neighbors >= 1000) break;
                            }
                        }
                    }



                }
            }
        }


        assert(num_neighbors > 5);

        // normalize histogram
        if (isfinite(shot[offset])){
            double hist_sum = 0;
            for (int i = 0; i < descLength_; ++i)
                hist_sum += shot[offset + i] * shot[offset + i];
            float norm_fact = static_cast<float>(sqrt(hist_sum));

            for (int i = 0; i < descLength_; ++i)
                shot[offset + i] = shot[offset + i]/ norm_fact;
        }


    }

}



void SHOT352::computeDescriptor(pcl::PointCloud<pcl::SHOT352> &output, const Eigen::Vector4f &inv_radius,
        const Eigen::Vector4i &pc_dimension, const Eigen::Vector4i &min_pi, const Eigen::Vector4i &max_pi,
        const IndicesConstPtr &grid_start_indices, const IndicesConstPtr &grid_end_indices) {

    descLength_ = nr_grid_sector_ * (nr_shape_bins_ + 1);

    assert(descLength_ == 352);

    int _grid_count_max = pc_dimension[0] + pc_dimension[0] * pc_dimension[1] + pc_dimension[0] * pc_dimension[1] * pc_dimension[2];

    // compute local reference
//    pcl::PointCloud<pcl::ReferenceFrame> local_ref;

    // gpu implementation of lrf, local frame is not determinastic

//    SHOT_LRF lrf;
//    lrf.setRadius(_radius);
//    lrf.setInputCloud(_input);
//    lrf.setSurface(_surface);
//    lrf.setNormals(_normals);
//    lrf.setKeptIndices(_kept_indices);
//    lrf.compute(local_ref, inv_radius, pc_dimension, min_pi);

    std::cout << "-------------------calculating descriptor shot------------------------" << std::endl;

    std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();

    std::vector<int> indices;
    for (int i =0; i < _input->points.size();i++){
        indices.emplace_back(i);
    }
    IndicesPtr indices_ = boost::make_shared<std::vector<int>>(indices);

    pcl::SHOTLocalReferenceFrameEstimationOMP<PointType> lrf_estimator;
    pcl::PointCloud<pcl::ReferenceFrame>::Ptr default_frames(new pcl::PointCloud<pcl::ReferenceFrame>);
    lrf_estimator.setRadiusSearch (_radius);
    lrf_estimator.setInputCloud (_input);
    lrf_estimator.setSearchSurface(_surface);
    lrf_estimator.setIndices (indices_);
    lrf_estimator.compute (*default_frames);

    std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>( t2 - t1 ).count();
    std::cout << "local reference calculation takes: " << duration << std::endl;

    int N = static_cast<int> (_input->points.size());
    dim3 numThreadsPerBlock = (static_cast<u_int32_t >((N + blockSize - 1)/blockSize));

    int N_surface = static_cast<int>(_surface->points.size());

    int *dev_gridCellStartIndices = NULL;
    hipMalloc((void**)&dev_gridCellStartIndices, _grid_count_max * sizeof(int));
    checkCUDAError("hipMalloc dev_gridCellStartIndices failed");
    hipMemcpy(dev_gridCellStartIndices, &(*grid_start_indices)[0], sizeof(int) * _grid_count_max, hipMemcpyHostToDevice);
    checkCUDAError("cudaMmcpy dev_gridCellStartIndices failed");

    int *dev_gridCellEndIndices = NULL;
    hipMalloc((void**)&dev_gridCellEndIndices, _grid_count_max * sizeof(int));
    checkCUDAError("hipMalloc dev_gridCellEndIndices failed");
    hipMemcpy(dev_gridCellEndIndices, &(*grid_end_indices)[0], sizeof(int) * _grid_count_max, hipMemcpyHostToDevice);
    checkCUDAError("hipMemcpy dev_gridCellEndIndices failed");

    int *dev_array_indices = NULL;
    hipMalloc((void**)&dev_array_indices, N_surface * sizeof(int));
    checkCUDAError("dev_grid malloc");
    hipMemcpy(dev_array_indices, &(*_array_indices)[0], N_surface * sizeof(int), hipMemcpyHostToDevice);
    checkCUDAError("dev_grid memcpy");


    int *dev_kept_indices = NULL;
    hipMalloc((void**)&dev_kept_indices, N * sizeof(int));
    checkCUDAError("cuda malloc kept indices error");
    hipMemcpy(dev_kept_indices, &(*_kept_indices)[0], N * sizeof(int), hipMemcpyHostToDevice);
    checkCUDAError("cuda memcpy kept_indices error");

    pcl::ReferenceFrame *dev_lrf = NULL;
    hipMalloc((void**)&dev_lrf, N * sizeof (pcl::ReferenceFrame));
    checkCUDAError("cuda malloc dev_lrf error");

    hipMemcpy(dev_lrf, &default_frames->points[0], N * sizeof(pcl::ReferenceFrame), hipMemcpyHostToDevice);
    checkCUDAError("cuda Memcpy lrf error");

    pcl::Normal *dev_normals = NULL;
    hipMalloc((void**)&dev_normals, sizeof(pcl::Normal) * N_surface);
    checkCUDAError("cuda malloc dev_normals error");
    hipMemcpy(dev_normals, &_normals->points[0], N_surface * sizeof(pcl::Normal), hipMemcpyHostToDevice);
    checkCUDAError("cuda memcpy dev_normals error");

    PointType* dev_pos_surface;
    hipMalloc((void**)&dev_pos_surface, N_surface * sizeof(PointType));
    checkCUDAError("malloc dps error");
    hipMemcpy(dev_pos_surface, &(_surface->points[0]), N_surface * sizeof(PointType), hipMemcpyHostToDevice);
    checkCUDAError("memcpy ps error");

    float *dev_shot = NULL;
    hipMalloc((void**)&dev_shot, N * descLength_ * sizeof(float));
    checkCUDAError("dev_bin_dist error");


    computeSHOT<<<numThreadsPerBlock, blockSize>>> (N, dev_pos_surface, _radius, dev_kept_indices, dev_array_indices,
             dev_gridCellStartIndices, dev_gridCellEndIndices, pc_dimension, min_pi, max_pi, inv_radius,
            dev_normals, dev_lrf, nr_shape_bins_, dev_shot);
    checkCUDAError("compute shot error");

//    std::vector<Eigen::Matrix4f, Eigen::aligned_allocator<Eigen::Matrix4f> > rototranslations;

    std::vector<float> shot(static_cast<uint32_t >(N * descLength_));
    hipMemcpy(&shot[0], dev_shot, sizeof(float) * N * descLength_, hipMemcpyDeviceToHost);
    checkCUDAError("copy shot error");


    // write to output
    for (int i = 0; i < N; ++i){
        int offset = i * descLength_;
        if (!isfinite(shot[offset])){
            output.is_dense = false;
            for (int j = 0; j < descLength_; ++j)
                output.points[i].descriptor[j] = std::numeric_limits<float >::quiet_NaN();
            for (int j = 0; j < 9; ++j)
                output.points[i].rf[j] = std::numeric_limits<float >::quiet_NaN();
        }
        else{
            for (int j = 0; j < descLength_; ++j)
                output.points[i].descriptor[j] = shot[offset + j];
            for (int j = 0; j < 3; ++j){
                output.points[i].rf[j] = default_frames->points[i].x_axis[j];
                output.points[i].rf[j + 3] = default_frames->points[i].y_axis[j];
                output.points[i].rf[j + 6] = default_frames->points[i].z_axis[j];
            }
        }
    }

    std::cout << "descriptor calculation done" << std::endl;

    hipFree(dev_kept_indices);
    hipFree(dev_lrf);
    hipFree(dev_normals);
    hipFree(dev_pos_surface);
    hipFree(dev_shot);
    checkCUDAError("cudafree err");

}
