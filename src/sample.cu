#include "hip/hip_runtime.h"
#include "detection.h"
#define VERBOSE 1
#include <cstdio>
#include <thrust/copy.h>
#include <thrust/remove.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/device_ptr.h>
/*
 * Atomic functions for float
 * https://stackoverflow.com/questions/17399119/cant-we-use-atomic-operations-for-floating-point-variables-in-cuda
 */
__device__ static float atomicMax(float* address, float val)
{
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
                          __float_as_int(::fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

__device__ static float atomicMin(float* address, float val){
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
                          __float_as_int(::fminf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}



__global__ void getMinMax(int N ,const PointType *pts_in, Eigen::Vector4f *min_pt, Eigen::Vector4f  *max_pt){
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index < N){

        PointType pt = pts_in[index] ;
        if (isfinite(pt.x) && isfinite(pt.y) && isfinite(pt.z)){
            //float pt_x = pt.x, pt_y = pt.y, pt_z = pt.z;
            atomicMin(&(*min_pt)[0], pt.x);
            atomicMin(&(*min_pt)[1], pt.y);
            atomicMin(&(*min_pt)[2], pt.z);
            atomicMax(&(*max_pt)[0], pt.x);
            atomicMax(&(*max_pt)[1], pt.y);
            atomicMax(&(*max_pt)[2], pt.z);
        }
    }
}


__device__ int kernComputeIndices(Eigen::Vector4i pos, Eigen::Vector4i grid_res){
    return pos[0] + pos[1] * grid_res[0] + pos[2] * grid_res[1] * grid_res[2];
}

__global__ void kernComputeIndices(int N, Eigen::Vector4i grid_res, Eigen::Vector4i grid_min,
        Eigen::Vector4f inv_radius, PointType *pos, int *indices, int *grid_indices){
    int index = threadIdx.x + (blockIdx.x *blockDim.x);
    if (index < N){
        PointType pt = pos[index] ;
        if (isfinite(pt.x) && isfinite(pt.y) && isfinite(pt.z)){
            Eigen::Vector4i ijk(static_cast<int>(floor(pt.x * inv_radius[0])),
                                static_cast<int>(floor(pt.y * inv_radius[1])), static_cast<int>(floor(pt.z * inv_radius[2])), 0);


            Eigen::Vector4i offset = ijk - grid_min;
//            printf("offset is %d, %d, %d \n",offset[0], offset[1], offset[2]);
//            printf("grid res is %d, %d, %d \n", grid_res[0], grid_res[1], grid_res[2]);
            grid_indices[index] = kernComputeIndices(offset, grid_res);
//            printf("indice is %d \n", grid_indices[index] );
            indices[index] = index;
        }

    }
}


//Copy index
__global__ void isfirst_indices(int N, int *input, int *res) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        if (tid == 0) {
            res[0] = 0;
        } else if (input[tid] != input[tid - 1]) {
            res[tid] = tid;
        } else {
            res[tid] = -1;
        }
    }
}


struct isFirst {
    __host__ __device__ bool operator()(const int x) {
        return (x != -1);
    }
};


//__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
//                                         int *gridCellStartIndices, int *gridCellEndIndices) {
//    // Identify the start point of each cell in the gridIndices array.
//    // This is basically a parallel unrolling of a loop that goes
//    // "this index doesn't match the one before it, must be a new cell!"
//    int index = threadIdx.x + (blockIdx.x * blockDim.x);
//    if (index >= N) return;
//    // corner cases
//    if (index == 0) gridCellStartIndices[particleGridIndices[index]] = index;
//    else if (index == N - 1) gridCellEndIndices[particleGridIndices[index]] = index;
//
//    else if (particleGridIndices[index] != particleGridIndices[index + 1]){
//        gridCellEndIndices[particleGridIndices[index]] = index;
//        gridCellStartIndices[particleGridIndices[index + 1]] = index + 1;
//    }
//}
__device__ float kernComputeDist(PointType pos, Eigen::Vector4i ijk){
    return (pos.x - ijk[0]) * (pos.x - ijk[0]) + (pos.y - ijk[1]) * (pos.y - ijk[1])
            + (pos.z - ijk[2]) * (pos.z - ijk[2]);
}


// uniform downsampling the points
__global__ void kernComputeDist(int N, const PointType *pts_in, int *dist, Eigen::Vector4f inv_radius){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index < N){
        PointType pt = pts_in[index] ;
        if (isfinite(pt.x) && isfinite(pt.y) && isfinite(pt.z)){
            Eigen::Vector4i ijk(static_cast<int>(floor(pt.x * inv_radius[0])),
                                static_cast<int>(floor(pt.y * inv_radius[1])), static_cast<int>(floor(pt.z * inv_radius[2])), 0);
            dist[index] = kernComputeDist(pts_in[index], ijk);
        }
    }
}


__global__ void kernUniformDownSample(int N, PointType *pts_in, PointType *pts_out, int *indices){
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index < N){
        pts_out[index] = pts_in[indices[index]];
    }
}

UniformDownSample::UniformDownSample(float radius): radius(radius), N_new(0), N(0){
    hipMalloc((void**)&dev_min, sizeof(Eigen::Vector4f));
    hipMalloc((void**)&dev_max, sizeof(Eigen::Vector4f));
    checkCUDAError("hipMalloc min,max");
}

UniformDownSample::~UniformDownSample() {
    hipFree(dev_min);
    hipFree(dev_max);
    hipFree(dev_grid_indices);
    hipFree(dev_array_indices);
    hipFree(dev_new_pc);
    hipFree(dev_pc);
    hipFree(dev_tmp);
    dev_min = NULL;
    dev_max = NULL;
    dev_grid_indices = NULL;
    dev_array_indices = NULL;
    dev_new_pc = NULL;
    dev_tmp = NULL;
    dev_pc = NULL;
}

void UniformDownSample::setRadius(float radius) {this->radius = radius;}

void UniformDownSample::downSample(const pcl::PointCloud<PointType >::ConstPtr input,
        pcl::PointCloud<PointType>::Ptr output) {
    N = (int)(*input).size();
    dim3 fullBlockPerGrid_points ((N + blockSize - 1)/blockSize);
    hipMalloc((void**) &dev_pc, N * sizeof(PointType));
    hipMemcpy(dev_pc, &(*input).points[0], N * sizeof(PointType), hipMemcpyHostToDevice);
    checkCUDAError("hipMemcpy pc");

    // calculate min max for the pc

    Eigen::Vector4f min_p, max_p;

    min_p.setConstant(FLT_MAX);
    max_p.setConstant(-FLT_MAX);
    hipMemcpy(dev_min, &min_p, sizeof(Eigen::Vector4f), hipMemcpyHostToDevice);
    hipMemcpy(dev_max, &max_p, sizeof(Eigen::Vector4f), hipMemcpyHostToDevice);
    checkCUDAError("hipMemcpy min,max");

    getMinMax <<< fullBlockPerGrid_points, blockSize>>>(N, dev_pc, dev_min, dev_max);
    checkCUDAError("getMinMax error");
    hipMemcpy(&min_p, dev_min, sizeof(Eigen::Vector4f), hipMemcpyDeviceToHost);
    checkCUDAError("memcpy min  error");
    hipMemcpy(&max_p, dev_max, sizeof(Eigen::Vector4f), hipMemcpyDeviceToHost);
    checkCUDAError("memcpy max error");
    // device the pc into cells

    Eigen::Vector4f inv_radius = Eigen::Array4f::Ones()/ (Eigen::Vector4f(radius, radius, radius, 1.0f).array());
    Eigen::Vector4i max_pi(static_cast<int>(floor(max_p[0] * inv_radius[0])),
            static_cast<int>(floor(max_p[1] * inv_radius[1])), static_cast<int>(floor(max_p[2] * inv_radius[2])), 0);
    Eigen::Vector4i min_pi(static_cast<int>(floor(min_p[0] * inv_radius[0])),
            static_cast<int>(floor(min_p[1] * inv_radius[1])), static_cast<int>(floor(inv_radius[2] * min_p[2])), 0);


    Eigen::Vector4i pc_dimension = max_pi - min_pi + Eigen::Vector4i::Ones();
    pc_dimension[3] = 0;

    int total_grid_count = pc_dimension[0] * pc_dimension[1] * pc_dimension[2];

    //Eigen::Vector4i grid_res = Eigen::Vector4i(1, pc_dimension[0], pc_dimension[0] * pc_dimension[1], 0);

    hipMalloc((void**)&dev_grid_indices, N * sizeof(int));
    checkCUDAError("hipMalloc dev_indices error");
    hipMalloc((void**)&dev_array_indices, N * sizeof(int));
    checkCUDAError("hipMalloc dev_indices error");

    kernComputeIndices <<< fullBlockPerGrid_points, blockSize >>> (N, pc_dimension, min_pi,
        inv_radius, dev_pc, dev_array_indices, dev_grid_indices);
    checkCUDAError("kernComputeIndices Failed");

    thrust::device_ptr<int> dev_thrust_grid_indices =  thrust::device_ptr<int>(dev_grid_indices);
    thrust::device_ptr<int> dev_thrust_array_indices = thrust::device_ptr<int>(dev_array_indices);
    thrust::sort_by_key(dev_thrust_grid_indices, dev_thrust_grid_indices + N, dev_thrust_array_indices);

    // get the coherent val for original entry
    hipMalloc((void**)&dev_new_pc, N * sizeof(PointType));
    checkCUDAError("malloc dev_coherent_pc error");
    kernUniformDownSample <<<fullBlockPerGrid_points, blockSize >>> (N, dev_pc, dev_new_pc, dev_array_indices);
    checkCUDAError("kernGetCoherentVal Failed");

    // get the first occurance of unique indices
    hipMalloc((void**)&dev_tmp, N * sizeof(int));
    checkCUDAError("hipMalloc dev_tmp failed");

    isfirst_indices<<< fullBlockPerGrid_points, blockSize>>> (N, dev_grid_indices, dev_tmp);
    checkCUDAError("isfirst indices error");

    //thrust::device_ptr<int> dev_thrust_tmp = thrust::device_ptr<int>(dev_tmp);

    int * new_end = thrust::partition(thrust::device, dev_tmp, dev_tmp + N, isFirst());

    int num_unique = new_end - dev_tmp;
//
    std::vector<int>unique_indices(num_unique, 0);
    hipMemcpy(&unique_indices[0], dev_tmp, num_unique  * sizeof(int), hipMemcpyDeviceToHost);
    checkCUDAError("Memcpy device to host Failed");
//    hipMalloc((void**)&dev_dist, N * sizeof(int));
//    checkCUDAError("malloc dev_dist error");
    kernComputeDist <<< fullBlockPerGrid_points, blockSize >>>(N, dev_new_pc, dev_tmp, inv_radius);
    checkCUDAError("KernComputeDist failed");

    std::cout << "---------------------------------------------------------" << std::endl;
    (*output).height = 1;
    (*output).is_dense = true;
    (*output).width = static_cast<uint32_t>(num_unique);
    (*output).points.resize (static_cast<uint32_t>(num_unique));

    //for (auto & i : unique_indices) std::cout << i << std::endl;


    std::vector<int>dist(N, 0);
    std::vector<int>indices(N, 0);
    hipMemcpy(&dist[0], dev_tmp, N * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&indices[0], dev_array_indices, N * sizeof(int), hipMemcpyDeviceToHost);

    int cell_start = unique_indices[0];
    int cell_end = unique_indices[1];
    for (int i = 2; i < unique_indices.size(); i++){
       std::sort(indices.begin() + cell_start, indices.begin() + cell_end, [&](const int& a, const int& b) {
           return (dist[a] < dist[b]);
       });

       //std::cout << "min is " << indices[cell_start] << std::endl;
       (*output).points[i] = (*input).points[indices[cell_start]];
       cell_start = cell_end;
       cell_end = unique_indices[i];
    }
    (*output).points[unique_indices.size()-1] = (*input).points[unique_indices[unique_indices.size()- 1]];

//
//
//
//    // sort inplace
//    thrust::sort_by_key(dev_thrust_dist, dev_thrust_dist + N, dev_thrust_array_indices);
//    kernUniformDownSample <<<fullBlockPerGrid_points, blockSize >>> (N, dev_pc, dev_new_pc, dev_array_indices);
//    checkCUDAError("kernGetCoherentVal Failed");

//    std::vector<PointType>new_pts(num_unique);
//    hipMemcpy(&new_pts[0], dev_new_pc, num_unique * sizeof(PointType), hipMemcpyDeviceToHost);




    //thrust::copy(dev_tmp, new_end, unique_indices.begin());



//        hipMalloc((void**)&dev_grid_start, total_grid_count * sizeof(int));
//        hipMalloc((void**)&dev_grid_end, total_grid_count * sizeof(int))
//        hipMemset(dev_grid_start, -1, total_grid_count * sizeof(int));
//        hipMemset(dev_grid_end, -1, total_grid_count * sizeof(int));
//
//        kernIdentifyCellStartEnd <<<fullBlockPerGrid_points, blockSize >>> (N, dev_grid_indices,
//                dev_grid_start, dev_grid_end);
//        checkCUDAErrorWithLine("kernIdentifyCellStartEnd Failed");



    // this is rather unefficeint
    //hipMalloc((void**)&dev_distance, N * sizeof(int));
//    std::vector<int> grid_indices(N, 0);
//    hipMemcpy(&grid_indices[0], dev_grid_indices, N  * sizeof(int), hipMemcpyDeviceToHost);
//    for (auto i:grid_indices)
//        std::cout << i << std::endl;
//    checkCUDAError("Memcpy device to host Failed");


//    thrust::counting_iterator
//    for (int i = 0; i < total_grid_count; i++){
//
//    }


//
    //for (int i = 0; i < 100; i++) std::cout << grid_indices[i] << ",";



#if VERBOSE
    std::cout << "---------------------------------------------------------" << std::endl;
    std::cout << "Num unique element is " << num_unique << std::endl;
    std::cout << "---------------------------------------------------------" << std::endl;
    std::cout << "Min is " << min_pi << std::endl;
    std::cout << "---------------------------------------------------------" << std::endl;
    std::cout << "Max is " << max_pi << std::endl;
    std::cout << "---------------------------------------------------------" << std::endl;
    std::cout << "The inverse radius is " << inv_radius << std::endl;
    std::cout << "---------------------------------------------------------" << std::endl;
    std::cout << "The grid count is " << total_grid_count << std::endl;
    std::cout << "---------------------------------------------------------" << std::endl;
    std::cout << "The point cloud dimension is " << pc_dimension << std::endl;
    std::cout << "---------------------------------------------------------" << std::endl;

#endif

    //kernUniformDownSample <<<fullBlockPerGrid_points, blockSize>>> (n_model, radius, dev_ss_pc_model, dev_ss_pc_model);
}
