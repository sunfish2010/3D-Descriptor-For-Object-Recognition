#include "hip/hip_runtime.h"

#include "grid.h"

static hipEvent_t start, stop;
/*
 * Atomic functions for float
 * https://stackoverflow.com/questions/17399119/cant-we-use-atomic-operations-for-floating-point-variables-in-cuda
 */
__device__ static float atomicMax(float* address, float val)
{
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
                          __float_as_int(::fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

__device__ static float atomicMin(float* address, float val){
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
                          __float_as_int(::fminf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}


/** \brief get min max for the point cloud  **/
__global__ void getMinMax(int N ,const PointType *pts_in, Eigen::Vector4f *min_pt, Eigen::Vector4f  *max_pt){
    __shared__ float min_max[6];
    for (int i = 0; i < 3; ++i)
        min_max[i] = FLT_MAX;
    for (int i = 3; i < 6; ++i)
        min_max[i] = FLT_MIN;
    __syncthreads();
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index < N){
        PointType pt = pts_in[index] ;
        if (isfinite(pt.x) && isfinite(pt.y) && isfinite(pt.z)){
            atomicMin(&min_max[0], pt.x);
            atomicMin(&min_max[1], pt.y);
            atomicMin(&min_max[2], pt.z);
            atomicMax(&min_max[3], pt.x);
            atomicMax(&min_max[4], pt.y);
            atomicMax(&min_max[5], pt.z);
        }
    }
    __syncthreads();
    atomicMin(&(*min_pt)[0], min_max[0]);
    atomicMin(&(*min_pt)[1], min_max[1]);
    atomicMin(&(*min_pt)[2], min_max[2]);
    atomicMax(&(*max_pt)[0], min_max[3]);
    atomicMax(&(*max_pt)[1], min_max[4]);
    atomicMax(&(*max_pt)[2], min_max[5]);
}

///** \brief 3D to 1D indice  **/
//__device__ int kernComputeIndices(Eigen::Vector4i pos, Eigen::Vector4i grid_res){
//    return
//}

/** \brief compute the indices the pt belongs to  **/
__global__ void kernComputeIndices(int N, Eigen::Vector4i grid_res, Eigen::Vector4i grid_min,
                                   Eigen::Vector4f inv_radius, PointType *pos, int *indices, int *grid_indices){
    int index = threadIdx.x + (blockIdx.x *blockDim.x);
    if (index < N){
        PointType pt = pos[index] ;
        if (isfinite(pt.x) && isfinite(pt.y) && isfinite(pt.z)){
            Eigen::Vector4i ijk(static_cast<int>(floor(pt.x * inv_radius[0])),
                                static_cast<int>(floor(pt.y * inv_radius[1])), static_cast<int>(floor(pt.z * inv_radius[2])), 0);


            Eigen::Vector4i offset = ijk - grid_min;
            int idx = offset[0] + offset[1] * grid_res[0] + offset[2] * grid_res[1] * grid_res[2];
            grid_indices[index] = idx;
            indices[index] = index;
        }

    }
}


/** \brief compute start and end indice of each grid  **/
__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
                                         int *gridCellStartIndices, int *gridCellEndIndices) {
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index < N) {
        // corner cases
        if (index == 0) gridCellStartIndices[particleGridIndices[index]] = index;
        else if (index == N - 1) gridCellEndIndices[particleGridIndices[index]] = index;

        else if (particleGridIndices[index] != particleGridIndices[index + 1]){
            gridCellEndIndices[particleGridIndices[index]] = index;
            gridCellStartIndices[particleGridIndices[index + 1]] = index + 1;
        }
    }

}


/** \brief compute point cloud properties for later use  **/
void Grid::computeSceneProperty(const pcl::PointCloud<PointType>::ConstPtr &input, const IndicesPtr &grid_indices,
        const IndicesPtr &array_indices, Eigen::Vector4f &inv_radius, Eigen::Vector4i &pc_dimension,
        Eigen::Vector4i &min_pi,  Eigen::Vector4i &max_pi) {
    if (!input || radius <= 0 || !grid_indices || !array_indices){
        std::cerr <<  "ComputeSceneProperty input not set correctly " << std::endl;
    }
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float miliseconds = 0;
    PointType *dev_pc=NULL;
    int *dev_grid_indices=NULL;
    int *dev_array_indices=NULL;

    N = (int)(*input).size();
    dim3 fullBlockPerGrid_points (static_cast<u_int32_t >((N + blockSize - 1)/blockSize));

    hipEventRecord(start);
    hipMalloc((void**) &dev_pc, N * sizeof(PointType));
    checkCUDAError("hipMalloc pc error");
    hipMemcpy(dev_pc, &(*input).points[0], N * sizeof(PointType), hipMemcpyHostToDevice);
    checkCUDAError("hipMemcpy pc");
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&miliseconds, start, stop);
    std::cout << "allocate and memcpy pt takes: " << miliseconds << std::endl;
    // calculate min max for the pc

    // after timing, cpu much faster so just use cpu version
    Eigen::Vector4f min_p, max_p;
    pcl::getMinMax3D<PointType>(*input, min_p, max_p);
//
//    min_p.setConstant(FLT_MAX);
//    max_p.setConstant(-FLT_MAX);
//    hipEventRecord(start);
//    hipMalloc((void**)&dev_min, sizeof(Eigen::Vector4f));
//    checkCUDAError("hipMalloc min");
//    hipMalloc((void**)&dev_max, sizeof(Eigen::Vector4f));
//    checkCUDAError("hipMalloc max");
//    hipMemcpy(dev_min, &min_p, sizeof(Eigen::Vector4f), hipMemcpyHostToDevice);
//    hipMemcpy(dev_max, &max_p, sizeof(Eigen::Vector4f), hipMemcpyHostToDevice);
//    checkCUDAError("hipMemcpy min,max");
//
//    getMinMax <<< fullBlockPerGrid_points, blockSize>>>(N, dev_pc, dev_min, dev_max);
//    checkCUDAError("getMinMax error");
//    hipMemcpy(&min_p, dev_min, sizeof(Eigen::Vector4f), hipMemcpyDeviceToHost);
//    checkCUDAError("memcpy min  error");
//    hipMemcpy(&max_p, dev_max, sizeof(Eigen::Vector4f), hipMemcpyDeviceToHost);
//    checkCUDAError("memcpy max error");
//    hipEventRecord(stop);
//    hipEventSynchronize(stop);
//    hipEventElapsedTime(&miliseconds, start, stop);
//    std::cout << "calculating min max takes  " << miliseconds << std::endl;
    // device the pc into cells

    inv_radius = Eigen::Array4f::Ones()/ (Eigen::Vector4f(radius, radius, radius, 1.0f).array());
    max_pi = Eigen::Vector4i(static_cast<int>(floor(max_p[0] * inv_radius[0])),
                           static_cast<int>(floor(max_p[1] * inv_radius[1])), static_cast<int>(floor(max_p[2] * inv_radius[2])), 0);
    min_pi = Eigen::Vector4i (static_cast<int>(floor(min_p[0] * inv_radius[0])),
                           static_cast<int>(floor(min_p[1] * inv_radius[1])), static_cast<int>(floor(inv_radius[2] * min_p[2])), 0);


    pc_dimension = max_pi - min_pi + Eigen::Vector4i::Ones();
    pc_dimension[3] = 0;

    hipEventRecord(start);
    hipMalloc((void**)&dev_grid_indices, N * sizeof(int));
    checkCUDAError("hipMalloc dev_indices error");
    hipMalloc((void**)&dev_array_indices, N * sizeof(int));
    checkCUDAError("hipMalloc dev_indices error");

    kernComputeIndices <<< fullBlockPerGrid_points, blockSize >>> (N, pc_dimension, min_pi,
            inv_radius, dev_pc, dev_array_indices, dev_grid_indices);
    checkCUDAError("kernComputeIndices Failed");

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&miliseconds, start, stop);
    std::cout << "calculating array & grid indices takes  " <<miliseconds << std::endl;



    // copy the results if needed.
    if (grid_indices){
        hipMemcpy(&(*grid_indices)[0], dev_grid_indices, N * sizeof(int), hipMemcpyDeviceToHost);
        checkCUDAError("kernCopy grid indices failed");
    }

    thrust::device_ptr<int> dev_thrust_gridIndices =  thrust::device_ptr<int>(dev_grid_indices);
    thrust::device_ptr<int> dev_thrust_arrayIndices = thrust::device_ptr<int>(dev_array_indices);

    // sort inplace
    thrust::sort_by_key(dev_thrust_gridIndices, dev_thrust_gridIndices + N, dev_thrust_arrayIndices);
    checkCUDAError("cuda sort error");

    if (array_indices){
        hipMemcpy(&(*array_indices)[0], dev_array_indices, N * sizeof(int), hipMemcpyDeviceToHost);
        checkCUDAError("kernCopy array indices failed");
    }

    hipFree(dev_array_indices);
    hipFree(dev_pc);
    checkCUDAError("cuda free error");


    int _grid_count = pc_dimension[0] * pc_dimension[1] * pc_dimension[2];

    int *dev_gridCellStartIndices = NULL;
    hipMalloc((void**)&dev_gridCellStartIndices, _grid_count * sizeof(int));
    checkCUDAError("hipMalloc dev_gridCellStartIndices failed");
    hipMemset(dev_gridCellStartIndices, -1, _grid_count * sizeof(int) );

    int *dev_gridCellEndIndices = NULL;
    hipMalloc((void**)&dev_gridCellEndIndices, _grid_count * sizeof(int));
    checkCUDAError("hipMalloc dev_gridCellStartIndices failed");

    kernIdentifyCellStartEnd <<<fullBlockPerGrid_points, blockSize >>> (N, dev_grid_indices,
            dev_gridCellStartIndices, dev_gridCellEndIndices);
    checkCUDAError("kernIdentifyCellStartEnd Failed");

    cellStartIndices.resize(_grid_count);
    hipMemcpy(&cellStartIndices[0], dev_gridCellStartIndices, sizeof(int) * _grid_count, hipMemcpyDeviceToHost);
    checkCUDAError("cell start");
    cellEndIndices.resize(_grid_count);
    hipMemcpy(&cellEndIndices[0], dev_gridCellEndIndices, sizeof(int) * _grid_count, hipMemcpyDeviceToHost);
    checkCUDAError("cell end");



//    checkCUDAError("cuda free error");
    hipFree(dev_grid_indices);
    hipFree(dev_gridCellEndIndices);
    hipFree(dev_gridCellStartIndices);
    checkCUDAError("cuda free error");

}


