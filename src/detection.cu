#include "detection.h"

//static int n_scene;
//static int n_model;

static PointType *dev_ss_pc_scene = NULL;
static PointType *dev_ss_pc_model = NULL;
static PointType *dev_kp_scene = NULL;
static PointType *dev_kp_model = NULL;



void detectionInit(const pcl::PointCloud<PointType >::ConstPtr model){
//    n_model =
    UniformDownSample filter = UniformDownSample(0.01);
   // pcl::PointCloud<PointType>::Ptr model_ds(new pcl::PointCloud<PointType>);
    filter.downSample(model);
    std::cout<< "Valid " << std::endl;

}

void detectFree(){
//    hipFree(dev_ss_pc_model);
//    hipFree(dev_ss_pc_scene);
//    hipFree(dev_kp_model);
//    hipFree(dev_kp_scene);

    dev_ss_pc_scene = NULL;
    dev_ss_pc_model = NULL;
    dev_kp_scene = NULL;
    dev_kp_model = NULL;

    checkCUDAError("cuda Free error");

}