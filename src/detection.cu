#include "detection.h"

//static int n_scene;
//static int n_model;

//static PointType *dev_ss_pc_scene = NULL;
//static PointType *dev_ss_pc_model = NULL;
//static PointType *dev_kp_scene = NULL;
//static PointType *dev_kp_model = NULL;


void detectionInit(pcl::PointCloud<PointType >::ConstPtr model,
                   pcl::PointCloud<PointType >::Ptr model_keypoints,
                   pcl::PointCloud<pcl::Normal>::ConstPtr model_normals,
                   pcl::PointCloud<pcl::SHOT352>::Ptr model_descriptors){

    //compute the common characters for the whole background
    unsigned int N = model->points.size();
    std::cout << "Num of pts is " << N << std::endl;
    float grid_res =  N > 300000? 0.03f:0.01f;
    IndicesPtr grid_indices(new std::vector<int>(N));
    IndicesPtr array_indices(new std::vector<int>(N));
//    std::vector<int> kept_indices;
    Grid grid;
    grid.setRadius(grid_res);
    grid.computeSceneProperty(model, grid_indices, array_indices);
    Eigen::Vector4i pc_dimension = grid.getDimension();
    Eigen::Vector4f inv_radius = grid.getInverseRadius();
    Eigen::Vector4i min_pi = grid.getSceneMin();

    std::cout << "---------------------------------------------------------" << std::endl;
    std::cout << "Min is " << min_pi << std::endl;
    std::cout << "---------------------------------------------------------" << std::endl;
    std::cout << "pc_dimension is " << pc_dimension << std::endl;
    std::cout << "---------------------------------------------------------" << std::endl;
    std::cout << "The inverse radius is " << inv_radius << std::endl;


    UniformDownSample filter;
//
//
//    filter.setKeptIndicesPtr(kept_indices);
    std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
//    filter.downSample(model, model_keypoints, grid_indices, array_indices, inv_radius);
////    filter.randDownSample(model, model_keypoints);
    filter.downSampleAtomic(model, inv_radius, pc_dimension, min_pi);
//    filter.display(model, model_keypoints);
    std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>( t2 - t1 ).count();
    std::cout << "GPU implementation  downsampling takes: " << duration << std::endl;


//
//    Eigen::Vector4i pc_dimension;
//
//    SHOT_LRF lrf;
//    lrf.setRadius(0.02f);
//    lrf.setInputCloud(model_keypoints);
//    lrf.setSurface(model);
//    lrf.setNormals(model_normals);

    IndicesConstPtr kept_indices = filter.getKeptIndice();
    SHOT descrip_shot;
    descrip_shot.setRadius(0.02);
    descrip_shot.setNormals(model_normals);
    descrip_shot.setInputCloud(model_keypoints);

    descrip_shot.setKeptIndices(kept_indices);
//    descrip_shot.setGridIndices(grid_indices);
    descrip_shot.setSurface(model);
    //descrip_shot.compute(*model_descriptors);


}

void detectFree(){
//    hipFree(dev_ss_pc_model);
//    hipFree(dev_ss_pc_scene);
//    hipFree(dev_kp_model);
//    hipFree(dev_kp_scene);

//    dev_ss_pc_scene = NULL;
//    dev_ss_pc_model = NULL;
//    dev_kp_scene = NULL;
//    dev_kp_model = NULL;
//
//    checkCUDAError("cuda Free error");

}